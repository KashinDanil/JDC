#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <cmath>
#include <unistd.h>
#include <chrono>
#include <thread>


template <class T>
class ArrayHost{
    T * values;
    long elem_numb;
public:
    explicit ArrayHost(long n):elem_numb(n) {
        values = new T[elem_numb];
    }
    void Init(void){
        time_t t;
        srand((unsigned) time(&t));
        for (int i = 0; i < elem_numb; i++) {
            values[i] = (T) (rand() & 0xFF) / 10.0f;
        }
    }
    T* get_values(void){
        return values;
    }
    long get_size(){
        return sizeof(T)*elem_numb;
    }
    ~ArrayHost(){
        delete []values;
    }
};


template <class T>
class ArrayDevice{
    T * values;
    long elem_numb;
public:
    explicit ArrayDevice(long n):elem_numb(n){
        hipMalloc((T**)&values,elem_numb*sizeof(T));


    };
    T* get_values(void){
        return values;
    }
    ~ArrayDevice(){
        hipFree(values);
    }
};


template <typename type>
int sumResult(type *device_array,const long array_size){
    int sum = 0;
    for (long i=0;i<array_size;i++){
        sum += device_array[i];
    }

    return sum;
}

template <typename type>
__global__ void sum_on_device(type *A, type *B, type *C) {
    int i = threadIdx.x  + blockIdx.x*blockDim.x;
    C[i] = A[i] + B[i];
}


int main(int argc, char** argv){

    using namespace std;
    
    long values_num = static_cast<long>(pow(2, 29)) * 1.5;
    if (argc < 3) {
        printf("Pass wished utilization in the first argument and duration in seconds in the second argument\n");
    }
    int wished_util = atoi(argv[1]);
    int duration = atoi(argv[2]);//duration in seconds

    ArrayHost<int> A_h (values_num);
    ArrayHost<int> B_h (values_num);

    A_h.Init();
    B_h.Init();

    ArrayHost<int> HostSum(values_num);
    ArrayHost<int> Device_to_HostSum(values_num);

    int max_threads_for_block = 1024;
    dim3 block(max_threads_for_block);
    dim3 grid ((values_num + max_threads_for_block - 1)/ max_threads_for_block);


    std::cout << "Trying to allocate " << values_num * 3 * sizeof(int) / pow(10, 9) << " GB on device" << std::endl;
    ArrayDevice<int> A_d(values_num);
    ArrayDevice<int> B_d(values_num);
    ArrayDevice<int> C_d(values_num);

    hipMemcpy(A_d.get_values(),A_h.get_values(),A_h.get_size(),hipMemcpyHostToDevice);
    hipMemcpy(B_d.get_values(),B_h.get_values(),B_h.get_size(),hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    std::cout << "Starting gpuload with utilization:" << wished_util << "%" << std::endl;
    auto start = chrono::steady_clock::now();
    typeof(start) endSleep;
    do {

        auto startSum = chrono::steady_clock::now();
        sum_on_device <int> <<<grid,block>>>(A_d.get_values(),B_d.get_values(),A_d.get_values());
        hipDeviceSynchronize();
        auto endSum = chrono::steady_clock::now();


        auto ms = chrono::duration_cast<chrono::milliseconds>(endSum - startSum).count();

        double time_diff = static_cast<double>(100 - wished_util)/static_cast<double>(wished_util);
        int time_sleep = static_cast<int>(ms * time_diff);
        auto startSleep = chrono::steady_clock::now();
        std::this_thread::sleep_for(std::chrono::milliseconds(time_sleep));
        endSleep = chrono::steady_clock::now();

    } while (duration > chrono::duration_cast<chrono::milliseconds>(endSleep - start).count() / 1000);

    hipMemcpy(Device_to_HostSum.get_values(),A_d.get_values(),A_h.get_size(),hipMemcpyDeviceToHost);

    int sum = sumResult(Device_to_HostSum.get_values(),values_num);

    return sum + 0 - sum; /* suppress unused variable warning */
};