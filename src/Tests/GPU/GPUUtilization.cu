#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <cmath>
#include <unistd.h>
#include <chrono>
#include <thread>

double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);

}


template <class T>
class ArrayHost{
    T * values;
    long elem_numb;
public:
    explicit ArrayHost(long n):elem_numb(n) {
        values = new T[elem_numb];
    }
    void Init(void){
        time_t t;
        srand((unsigned) time(&t));
        for (int i = 0; i < elem_numb; i++) {
            values[i] = (T) (rand() & 0xFF) / 10.0f;
        }
    }
    T* get_values(void){
        return values;
    }
    long get_size(){
        return sizeof(T)*elem_numb;
    }
    ~ArrayHost(){
        delete []values;
    }
};


template <class T>
class ArrayDevice{
    T * values;
    long elem_numb;
public:
    explicit ArrayDevice(long n):elem_numb(n){
        hipMalloc((T**)&values,elem_numb*sizeof(T));


    };
    T* get_values(void){
        return values;
    }
    ~ArrayDevice(){
        hipFree(values);
    }
};


template <typename type>
void checkResult(type *host_array,type *device_array,const long array_size){
    double epsilon  = 0.1;
    for (long i=0;i<array_size;i++){
        if(abs(device_array[i] - host_array[i]) > epsilon){
            std::cout<<"ERROR in position"<<i;
            break;
        }
    }
    std::cout<<"SUCCESS\n";
}

template <typename type>
void sum_on_host(type *A,type *B, type *C, long N, int iter_limit){
    for(long i =0;i<N;i++){
        C[i] = A[i] + iter_limit* B[i];
    }
}

template <typename type>
__global__ void sum_on_device(type *A, type *B, type *C) {
    int i = threadIdx.x  + blockIdx.x*blockDim.x;
    C[i] = A[i] + B[i];
}


int main(int argc, char** argv){

    using namespace std;
    
    long values_num = static_cast<long>(pow(2, 29)) * 1.5;
    if (argc < 3) {
        printf("Pass wished utilization in the first argument and iterations limit in the second argument\n");
    }
    int wished_util = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);

    ArrayHost<int> A_h (values_num);
    ArrayHost<int> B_h (values_num);

    A_h.Init();
    B_h.Init();

    ArrayHost<int> HostSum(values_num);
    ArrayHost<int> Device_to_HostSum(values_num);

    int max_threads_for_block = 1024;
    dim3 block(max_threads_for_block);
    dim3 grid ((values_num + max_threads_for_block - 1)/ max_threads_for_block);

    double start_on_host = cpuSecond();
    sum_on_host<int>(A_h.get_values(),B_h.get_values(),HostSum.get_values(),values_num, iter_limit);
    double end_on_host = cpuSecond();


    std::cout << "Trying to allocate " << values_num * 3 * sizeof(int) / pow(10, 9) << " GB on device" << std::endl;
    ArrayDevice<int> A_d(values_num);
    ArrayDevice<int> B_d(values_num);
    ArrayDevice<int> C_d(values_num);

    hipMemcpy(A_d.get_values(),A_h.get_values(),A_h.get_size(),hipMemcpyHostToDevice);
    hipMemcpy(B_d.get_values(),B_h.get_values(),B_h.get_size(),hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    std::cout << "Starting gpuload with utilization:" << wished_util << "%" << std::endl;
    for (int iter = 0; iter < iter_limit; iter++) {

        auto start = chrono::steady_clock::now();
        sum_on_device <int> <<<grid,block>>>(A_d.get_values(),B_d.get_values(),A_d.get_values());
        hipDeviceSynchronize();
        auto end = chrono::steady_clock::now();


        auto ms = chrono::duration_cast<chrono::milliseconds>(end - start).count();

        double time_diff = static_cast<double>(100 - wished_util)/static_cast<double>(wished_util);
        int time_sleep = static_cast<int>(ms * time_diff);
        auto start1 = chrono::steady_clock::now();
        std::this_thread::sleep_for(std::chrono::milliseconds(time_sleep));
        auto end1 = chrono::steady_clock::now();

//        std::cout << "Kernel time: " << ms << " , Sleep time: " << chrono::duration_cast<chrono::milliseconds>(end1 - start1).count() << std::endl;
    }

    hipMemcpy(Device_to_HostSum.get_values(),A_d.get_values(),A_h.get_size(),hipMemcpyDeviceToHost);

    checkResult(HostSum.get_values(),Device_to_HostSum.get_values(),values_num);
};